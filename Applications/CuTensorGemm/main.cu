#include <cassert>
#include <hip/hip_runtime.h>

#include <hiptensor.h>
#include <iostream>
#include <vector>

#define CHECK_CUDA(err)                                                        \
  do {                                                                         \
    if (err != hipSuccess) {                                                  \
      std::cerr << "CUDA error: " << hipGetErrorString(err) << " at "         \
                << __FILE__ << ":" << __LINE__ << std::endl;                   \
      exit(1);                                                                 \
    }                                                                          \
  } while (0)

#define CHECK_CUTENSOR(err)                                                    \
  do {                                                                         \
    if (err != HIPTENSOR_STATUS_SUCCESS) {                                      \
      std::cerr << "cuTENSOR error: " << hiptensorGetErrorString(err) << " at " \
                << __FILE__ << ":" << __LINE__ << std::endl;                   \
      exit(1);                                                                 \
    }                                                                          \
  } while (0)

int main() {
  // Print cuTENSOR version for debugging
  std::cout << "cuTENSOR version: " << cutensorGetVersion() << std::endl;

  // Create cuTENSOR handle
  hiptensorHandle_t handle;
  CHECK_CUTENSOR(hiptensorCreate(&handle));

  // Matrix dimensions (input [m,k], weights [k,n], output [m,n])
  int32_t m = 2; // batch size
  int32_t n = 3; // output features
  int32_t k = 4; // input features

  // Allocate device memory
  float *d_input, *d_weights, *d_output;
  CHECK_CUDA(hipMalloc(&d_input, m * k * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_weights, k * n * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_output, m * n * sizeof(float)));

  const uint32_t kAlignment = 128;
  assert(uintptr_t(d_input) % kAlignment == 0);
  assert(uintptr_t(d_weights) % kAlignment == 0);
  assert(uintptr_t(d_output) % kAlignment == 0);

  // TensorDescriptor
  hiptensorTensorDescriptor_t descA;
  std::vector<int64_t> extentA = {m, k}; // [m,k]
  CHECK_CUTENSOR(hiptensorCreateTensorDescriptor(handle, &descA, 2,
                                                extentA.data(), NULL, //
                                                HIPTENSOR_R_32F, kAlignment));
  hiptensorTensorDescriptor_t descB;
  std::vector<int64_t> extentB = {k, n}; // [k,n]
  CHECK_CUTENSOR(hiptensorCreateTensorDescriptor(handle, &descB, 2,
                                                extentB.data(), NULL, //
                                                HIPTENSOR_R_32F, kAlignment));
  hiptensorTensorDescriptor_t descC;
  std::vector<int64_t> extentC = {m, n}; // [m,n]
  CHECK_CUTENSOR(hiptensorCreateTensorDescriptor(handle, &descC, 2,
                                                extentC.data(), NULL, //
                                                HIPTENSOR_R_32F, kAlignment));

  // Initialize input data (same as original)
  std::vector<float> input = {1.0f, 2.0f, 3.0f, 4.0f,  //
                              1.0f, 2.0f, 3.0f, 4.0f}; // input [2,4]
  std::vector<float> weights = {
      1.0f, 0.0f, 0.0f, 1.0f, // w0
      0.0f, 1.0f, 1.0f, 0.0f, // w1
      1.0f, 1.0f, 1.0f, 1.0f  // [4,3]
  };
  std::vector<float> output(m * n);

  // Copy to device
  CHECK_CUDA(hipMemcpy(d_input, input.data(), m * k * sizeof(float),
                        hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_weights, weights.data(), k * n * sizeof(float),
                        hipMemcpyHostToDevice));

  // Set up contraction plan
  hiptensorOperationDescriptor_t opDesc;
  int32_t modeA[] = {0, 1}; // [m,k]
  int32_t modeB[] = {1, 2}; // [k,n]
  int32_t modeC[] = {0, 2}; // [m,n]
  const hiptensorComputeDescriptor_t descCompute = HIPTENSOR_COMPUTE_DESC_32F;
  CHECK_CUTENSOR(
      hiptensorCreateContraction(handle, &opDesc,                    //
                                descA, modeA, HIPTENSOR_OP_IDENTITY, //
                                descB, modeB, HIPTENSOR_OP_IDENTITY, //
                                descC, modeC, HIPTENSOR_OP_IDENTITY, //
                                descC, modeC, descCompute));

  // Create plan
  hiptensorPlanPreference_t planPref;
  const hiptensorAlgo_t algo = HIPTENSOR_ALGO_DEFAULT;
  CHECK_CUTENSOR(hiptensorCreatePlanPreference(handle, &planPref, algo,
                                              HIPTENSOR_JIT_MODE_NONE));

  // Query workspace estimate
  uint64_t workspaceSizeEstimate = 0;
  const hiptensorWorksizePreference_t workspacePref = HIPTENSOR_WORKSPACE_DEFAULT;
  CHECK_CUTENSOR(hiptensorEstimateWorkspaceSize(
      handle, opDesc, planPref, workspacePref, &workspaceSizeEstimate));

  // Allocate workspace
  void *d_workspace = nullptr;
  if (workspaceSizeEstimate > 0) {
    CHECK_CUDA(hipMalloc(&d_workspace, workspaceSizeEstimate));
  }

  hiptensorPlan_t plan;
  CHECK_CUTENSOR(hiptensorCreatePlan(handle, &plan, opDesc, planPref,
                                    workspaceSizeEstimate));

  // Execute contraction
  hipStream_t stream;
  float alpha = 1.0;
  float beta = 0.0;
  CHECK_CUDA(hipStreamCreate(&stream));
  CHECK_CUTENSOR(hiptensorContract(handle, plan,                       //
                                  (void *)&alpha, d_input, d_weights, //
                                  (void *)&beta, d_output, d_output,  //
                                  d_workspace, workspaceSizeEstimate, stream));

  // Synchronize stream before copying results
  CHECK_CUDA(hipStreamSynchronize(stream));

  // Copy result back to host
  CHECK_CUDA(hipMemcpy(output.data(), d_output, m * n * sizeof(float),
                        hipMemcpyDeviceToHost));

  // Print output
  std::cout << "Output: ";
  for (float val : output) {
    std::cout << val << " ";
  }
  std::cout << std::endl;

  // Cleanup
  CHECK_CUTENSOR(hiptensorDestroyPlan(plan));
  CHECK_CUTENSOR(hiptensorDestroyPlanPreference(planPref));
  CHECK_CUTENSOR(hiptensorDestroyOperationDescriptor(opDesc));
  CHECK_CUTENSOR(hiptensorDestroyTensorDescriptor(descA));
  CHECK_CUTENSOR(hiptensorDestroyTensorDescriptor(descB));
  CHECK_CUTENSOR(hiptensorDestroyTensorDescriptor(descC));
  CHECK_CUTENSOR(hiptensorDestroy(handle));
  CHECK_CUDA(hipStreamDestroy(stream));
  if (d_workspace)
    CHECK_CUDA(hipFree(d_workspace));
  CHECK_CUDA(hipFree(d_input));
  CHECK_CUDA(hipFree(d_weights));
  CHECK_CUDA(hipFree(d_output));

  return 0;
}